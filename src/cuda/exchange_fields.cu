#include "hip/hip_runtime.h"
#include "exchange_fields.hpp"

#include "atoms.hpp"
#include "vio.hpp"
#include "cuda_utils.hpp"
#include "internal.hpp"
#include "data.hpp"

#include <vector>


#include "cusp/array2d.h"
#include "cusp/coo_matrix.h"


int calculate_exchange_fields(int, int);

namespace cu = vcuda::internal;

namespace vcuda
{

   namespace internal
   {

      namespace exchange
      {


         bool exchange_initialised = false;

         bool J_isot_initialised = false;
         bool J_vect_initialised = false;
         bool J_tens_initialised = false;

         cu_real_array_t   spin3N;
         cu_real_array_t   field3N;

         cu_exch_mat_t  J_matrix_d;

         int initialise_exchange()
         {

            check_device_memory(__FILE__,__LINE__);

            spin3N.assign( 3*::atoms::num_atoms, 0);
            field3N.assign( 3*::atoms::num_atoms, 0);

            cusp::csr_matrix < int, cu::cu_real_t, cusp::host_memory > J_xx_matrix_h (
                  ::atoms::num_atoms,
                  ::atoms::num_atoms,
                  ::atoms::neighbour_list_array.size()
                  );

            cusp::csr_matrix < int, cu::cu_real_t, cusp::host_memory > J_yy_matrix_h (
                  ::atoms::num_atoms,
                  ::atoms::num_atoms,
                  ::atoms::neighbour_list_array.size()
                  );

            cusp::csr_matrix < int, cu::cu_real_t, cusp::host_memory > J_zz_matrix_h (
                  ::atoms::num_atoms,
                  ::atoms::num_atoms,
                  ::atoms::neighbour_list_array.size()
                  );


            J_xx_matrix_h.row_offsets[0] = 0.0;
            J_yy_matrix_h.row_offsets[0] = 0.0;
            J_zz_matrix_h.row_offsets[0] = 0.0;

            for (int atom = 0; atom < ::atoms::num_atoms; atom++) {
               J_xx_matrix_h.row_offsets[atom+1] = ::atoms::neighbour_list_end_index[atom]+1;
               J_yy_matrix_h.row_offsets[atom+1] = ::atoms::neighbour_list_end_index[atom]+1;
               J_zz_matrix_h.row_offsets[atom+1] = ::atoms::neighbour_list_end_index[atom]+1;
            }

            for (int i = 0; i < ::atoms::neighbour_list_array.size(); i++) {
               J_xx_matrix_h.column_indices[i] = ::atoms::neighbour_list_array[i];
               J_yy_matrix_h.column_indices[i] = ::atoms::neighbour_list_array[i];
               J_zz_matrix_h.column_indices[i] = ::atoms::neighbour_list_array[i];
            }

            cusp::array1d<int, cusp::host_memory> row_indices( J_xx_matrix_h.column_indices);
            cusp::array1d<int, cusp::host_memory> column_indices( J_xx_matrix_h.column_indices);
            cusp::offsets_to_indices( J_xx_matrix_h.row_offsets, row_indices);

            cusp::coo_matrix< int, cu::cu_real_t, cusp::host_memory> J_matrix_h(
                  3*::atoms::num_atoms,
                  3*::atoms::num_atoms,
                  3*::atoms::neighbour_list_array.size()
                  );

            //std::cerr << "Attempting to fill matrix." << std::endl;

            const int Natoms = ::atoms::num_atoms;
            const int Nnbrs = row_indices.size();
            for( int i = 0; i < Nnbrs; i++)
            {
               J_matrix_h.row_indices[i] = row_indices[i];
               J_matrix_h.row_indices[i+Nnbrs] = row_indices[i]+Natoms;
               J_matrix_h.row_indices[i+2*Nnbrs] = row_indices[i]+2*Natoms;

               J_matrix_h.column_indices[i] = column_indices[i];
               J_matrix_h.column_indices[i+Nnbrs] = column_indices[i]+Natoms;
               J_matrix_h.column_indices[i+2*Nnbrs] = column_indices[i]+2*Natoms;

               int iid = ::atoms::neighbour_interaction_type_array[i];
               J_matrix_h.values[i] = - ::atoms::i_exchange_list[iid].Jij;
               J_matrix_h.values[i+Nnbrs] = - ::atoms::i_exchange_list[iid].Jij;
               J_matrix_h.values[i+2*Nnbrs] = - ::atoms::i_exchange_list[iid].Jij;
            }

            zlog << zTs() << "Attempting matrix conversion from CSR to DIA now." << std::endl;

            cusp::convert( J_matrix_h, J_matrix_d);

            zlog << zTs() << "Matrix conversion complete." << std::endl;

            const size_t occupied_diagonals = count_diagonals(J_xx_matrix_h.num_rows, J_xx_matrix_h.num_rows, row_indices, J_xx_matrix_h.column_indices);
            const float size       = float(occupied_diagonals) * float(J_xx_matrix_h.num_rows);
            const float fill_ratio = size / std::max(1.0f, float(J_xx_matrix_h.num_entries));

            zlog << zTs() << "Cuda Matrix:\nDiagonals = " << occupied_diagonals << "\nsize = " << size << "\nfill ratio = "<< fill_ratio << std::endl;



            switch( ::atoms::exchange_type)
            {
               case 0: // Isotropic

                  //--------------------------------------------------------------
                  // Exchange is isotropic so Jxx = Jyy = Jzz
                  // and Jxy = Jxz = Jyx = 0
                  //--------------------------------------------------------------

                  for (int i = 0; i < ::atoms::neighbour_list_array.size(); i++) {
                     int iid = ::atoms::neighbour_interaction_type_array[i];
                     J_xx_matrix_h.values[i] = - ::atoms::i_exchange_list[iid].Jij;
                  }



                  cusp::convert(J_xx_matrix_h, J_xx_mat_d);

                  J_isot_initialised = true;

                  break;

               case 1: // Vector

                  //--------------------------------------------------------------
                  // Exchange is diagonal so Jxx != Jyy != Jzz
                  // and Jxy = Jxz = Jyx = 0
                  //--------------------------------------------------------------

                  // Copy J values from vampire exchange list to values list
                  for( int i = 0; i < ::atoms::neighbour_list_array.size(); i++)
                  {
                     int iid = ::atoms::neighbour_interaction_type_array[i];
                     J_xx_matrix_h.values[i] = - ::atoms::v_exchange_list[iid].Jij[0];
                     J_yy_matrix_h.values[i] = - ::atoms::v_exchange_list[iid].Jij[1];
                     J_zz_matrix_h.values[i] = - ::atoms::v_exchange_list[iid].Jij[2];
                  }

                  cusp::convert(J_xx_matrix_h, J_xx_mat_d);
                  cusp::convert(J_yy_matrix_h, J_yy_mat_d);
                  cusp::convert(J_zz_matrix_h, J_zz_mat_d);

                  J_vect_initialised = true;

                  break;

               case 2: // Tensor
                  std::cerr << "Error! Tensorial form of exchange not yet implemented in cuda version!" << std::endl;
                  zlog << zTs() << "Error! Tensorial form of exchange not yet implemented in cuda version!" << std::endl;
                  break;
            }

            exchange_initialised = true;

            //std::cout << "Made matrix" << std::endl;
            check_device_memory(__FILE__,__LINE__);
            check_cuda_errors(__FILE__,__LINE__);
            return EXIT_SUCCESS;
         }


         int finalise_exchange()
         {
            spin3N.cu_real_array_t::~cu_real_array_t();
            field3N.cu_real_array_t::~cu_real_array_t();
            J_matrix_d.cu_exch_mat_t::~cu_exch_mat_t ();
            check_cuda_errors(__FILE__,__LINE__);
            return EXIT_SUCCESS;
         }

         int calculate_exchange_fields()
         {

            check_cuda_errors(__FILE__,__LINE__);
              /*
            thrust::copy( cu::atoms::x_spin_array.begin(), cu::atoms::x_spin_array.end(), spin3N.begin());
            thrust::copy( cu::atoms::y_spin_array.begin(), cu::atoms::y_spin_array.end(), spin3N.begin() + ::atoms::num_atoms);
            thrust::copy( cu::atoms::z_spin_array.begin(), cu::atoms::z_spin_array.end(), spin3N.begin() + 2*::atoms::num_atoms);

            cusp::multiply(
                  J_matrix_d,
                  spin3N,
                  field3N);

            thrust::copy( field3N.begin(), field3N.begin() + ::atoms::num_atoms, cu::x_total_spin_field_array.begin() );
            thrust::copy( field3N.begin() + ::atoms::num_atoms, field3N.begin() + 2*::atoms::num_atoms, cu::y_total_spin_field_array.begin() );
            thrust::copy( field3N.begin() + 2*::atoms::num_atoms, field3N.end(), cu::z_total_spin_field_array.begin() );

              */
            switch( ::atoms::exchange_type)
            {
               case 0: // Isotropic

                  //--------------------------------------------------------------
                  // Exchange is isotropic so Jxx = Jyy = Jzz
                  // and Jxy = Jxz = Jyx = 0
                  //--------------------------------------------------------------

                  if( !exchange_initialised) initialise_exchange();

                  // FIXME This maybe boosted
                  // It should keep the old values stored in the spin field
                  // Since Jxx = Jyy = Jzz only the Jxx array is used

                  cusp::multiply(
                        J_xx_mat_d,
                        cu::atoms::x_spin_array,
                        cu::x_total_spin_field_array
                        );
                  cusp::multiply(
                        J_xx_mat_d,
                        cu::atoms::y_spin_array,
                        cu::y_total_spin_field_array
                        );
                  cusp::multiply(
                        J_xx_mat_d,
                        cu::atoms::z_spin_array,
                        cu::z_total_spin_field_array
                        );

                  break;

               case 1: // Vector exchange

                  //--------------------------------------------------------------
                  // Exchange is diagonal so Jxx != Jyy != Jzz
                  // and Jxy = Jxz = Jyx = 0
                  //--------------------------------------------------------------

                  if( !exchange_initialised) initialise_exchange();

                  cusp::multiply(
                        J_xx_mat_d,
                        cu::atoms::x_spin_array,
                        cu::x_total_spin_field_array
                        );
                  cusp::multiply(
                        J_yy_mat_d,
                        cu::atoms::y_spin_array,
                        cu::y_total_spin_field_array
                        );
                  cusp::multiply(
                        J_zz_mat_d,
                        cu::atoms::z_spin_array,
                        cu::z_total_spin_field_array
                        );

                  break;
            }

            check_cuda_errors(__FILE__,__LINE__);
            return EXIT_SUCCESS;
         }
      } // end namespace exchange

   } // end namespace internal

} // end namespace vcuda
