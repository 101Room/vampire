#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
//
//-----------------------------------------------------------------------------

// C++ standard library headers

// Vampire headers
#include "cuda.hpp"

// Local cuda headers
#include "internal.hpp"
#include "statistics.hpp"

#ifdef CUDA
namespace cu = vcuda::internal;
#endif

namespace vcuda{

#ifdef CUDA

   namespace stats
   {
         void update ()
         {

            cu::stats::__update_stat (
                  cu::stats::system_mask,
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization);

            cu::stats::__update_stat (
                  cu::stats::material_mask,
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization);

            cu::stats::__update_stat (
                  cu::stats::height_mask,
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization);

            cu::stats::__update_stat (
                  cu::stats::material_height_mask,
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization);



            // increase the counter
            cu::stats::counter++;

         }

         void get ()
         {

            cu::stats::__get_stat (
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization,
                  ::stats::system_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization,
                  ::stats::material_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization,
                  ::stats::height_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization,
                  ::stats::material_height_magnetization
                  );

         }

         void reset ()
         {
            cu::stats::counter = 0L;

            cu::stats::__reset_stat (
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization
                  );

         }

   } /* stats */

   namespace internal
   {
      namespace stats
      {


         void __update_stat (
               const IndexArray & mask,
               RealArray & stat,
               RealArray & mean_stat
               )
         {

            const int * d_mask = thrust::raw_pointer_cast (
                  mask.data());
            double * d_stat = thrust::raw_pointer_cast (
                  stat.data());
            double * d_accu = thrust::raw_pointer_cast (
                  mean_stat.data());

            double * d_x_spin = thrust::raw_pointer_cast(
                  cu::atoms::x_spin_array.data());
            double * d_y_spin = thrust::raw_pointer_cast(
                  cu::atoms::y_spin_array.data());
            double * d_z_spin = thrust::raw_pointer_cast(
                  cu::atoms::z_spin_array.data());
            double * d_spin_norm = thrust::raw_pointer_cast(
                  cu::atoms::spin_norm_array.data());

            int n_bins = stat.size ();
            int n_atoms = mask.size ();

            if (n_bins < 128)
            {

                // Use the shared memory implementation

               int n_bytes = 4 * stat.size() * sizeof(RealArray::value_type);
               hist_by_key_small_mask <<< cu::grid_size, cu::block_size, n_bytes >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
            }
            else
            {

               // Use the brute force implementation

               hist_by_key_big_mask <<< cu::grid_size, cu::block_size >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
            }


             // Reduce and accumulate

            int gs = n_bins / cu::block_size + 1;
            update_norm_and_accum <<< gs , cu::block_size >>> (
                  d_stat,
                  d_accu,
                  n_bins
                  );

         }


         void __get_stat (
               const RealArray& stat,
               const RealArray& mean_stat,
               ::stats::magnetization_statistic_t& local_stat
               )
         {

            /*
             * Copy to local arrays
             */

            thrust::host_vector<double> h_stat(stat.size());
            thrust::host_vector<double> h_mean_stat(mean_stat.size());

            thrust::copy(stat.begin(), stat.end(), h_stat.begin());
            thrust::copy(mean_stat.begin(), mean_stat.end(), h_mean_stat.begin());

            /*
             * Call the method in the magnetization_statistic_t instance
             */

            std::vector<double> stl_stat (h_stat.begin(), h_stat.end());
            std::vector<double> stl_mean_stat (h_mean_stat.begin(), h_mean_stat.end());

            local_stat.set_magnetization (
                  stl_stat,
                  stl_mean_stat,
                  counter);

         }


         void __reset_stat (
               RealArray& stat,
               RealArray& mean_stat
               )
         {
            thrust::fill(
                  stat.begin(),
                  stat.end(),
                  0.0);
            thrust::fill(
                  mean_stat.begin(),
                  mean_stat.end(),
                  0.0);
         }


         __global__ void hist_by_key_small_mask (
               const double * __restrict__ x_spin,
               const double * __restrict__ y_spin,
               const double * __restrict__ z_spin,
               const double * __restrict__ norm_spin,
               const int * __restrict__ mask,
               double * hist,
               int n_bins,
               int n_atoms
               )
         {
            extern __shared__ double block_hist[];

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Initialize block memory
                */
               block_hist[i] = 0.0;
            }

            __syncthreads ();

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {
               /*
                * Store stuff in the shared memory
                */
               int bin = mask[i];
               double mu_s = norm_spin[i];
               cu::atomicAdd (block_hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 3, mu_s);
            }

            __syncthreads ();

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Store stuff in the main memory
                */
               cu::atomicAdd (hist + 4 * i + 0, block_hist[4 * i + 0]);
               cu::atomicAdd (hist + 4 * i + 1, block_hist[4 * i + 1]);
               cu::atomicAdd (hist + 4 * i + 2, block_hist[4 * i + 2]);
               cu::atomicAdd (hist + 4 * i + 3, block_hist[4 * i + 3]);
            }

         }


         __global__ void hist_by_key_big_mask (
               const double * __restrict__ x_spin,
               const double * __restrict__ y_spin,
               const double * __restrict__ z_spin,
               const double * __restrict__ norm_spin,
               const int * __restrict__ mask,
               double * hist,
               int n_bins,
               int n_atoms
               )
         {

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {

                // Store stuff in the main memory

               int bin = mask[i];
               double mu_s = norm_spin[i];
               cu::atomicAdd (hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 3, mu_s);
            }
         }


         __global__ void update_norm_and_accum (
               double * hist,
               double * accum,
               int n_bins
               )
         {

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_bins;
                  i += blockDim.x * gridDim.x)
            {
               double mx = hist[4 * i + 0];
               double my = hist[4 * i + 1];
               double mz = hist[4 * i + 2];
               double ms = hist[4 * i + 3];

               double mm = sqrtf (
                     mx * mx +
                     my * my +
                     mz * mz
                     );

               hist[4 * i + 0] = mx / mm;
               hist[4 * i + 1] = my / mm;
               hist[4 * i + 2] = mz / mm;
               hist[4 * i + 3] = mm / ms;

               accum[4 * i + 0] += mx / mm;
               accum[4 * i + 1] += my / mm;
               accum[4 * i + 2] += mz / mm;
               accum[4 * i + 3] += mm / ms;
            }
         }

      } /* stats */
   } /* internal */

#endif

} // end of namespace cuda
