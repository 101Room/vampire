#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
//
//-----------------------------------------------------------------------------

// C++ standard library headers

// Vampire headers
#include "cuda.hpp"

// Local cuda headers
#include "cuda_utils.hpp"
#include "internal.hpp"
#include "statistics.hpp"

#ifdef CUDA
namespace cu = vcuda::internal;
#endif

namespace vcuda{

#ifdef CUDA

   namespace stats
   {
         void update ()
         {

            cu::stats::__update_stat (
                  cu::stats::system_mask,
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization);

            cu::stats::__update_stat (
                  cu::stats::material_mask,
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization);

            cu::stats::__update_stat (
                  cu::stats::height_mask,
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization);

            cu::stats::__update_stat (
                  cu::stats::material_height_mask,
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization);



            // increase the counter
            cu::stats::counter++;

         }

         void get ()
         {

            cu::stats::__get_stat (
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization,
                  ::stats::system_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization,
                  ::stats::material_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization,
                  ::stats::height_magnetization
                  );

            cu::stats::__get_stat (
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization,
                  ::stats::material_height_magnetization
                  );

         }

         void reset ()
         {
            cu::stats::counter = 0L;

            cu::stats::__reset_stat (
                  cu::stats::system_magnetization,
                  cu::stats::system_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::material_magnetization,
                  cu::stats::material_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::height_magnetization,
                  cu::stats::height_mean_magnetization
                  );

            cu::stats::__reset_stat (
                  cu::stats::material_height_magnetization,
                  cu::stats::material_height_mean_magnetization
                  );

         }

   } /* stats */

   namespace internal
   {
      namespace stats
      {


         void __update_stat (
               const cu_index_array_t & mask,
               cu_real_array_t & stat,
               cu_real_array_t & mean_stat
               )
         {

            const int * d_mask = thrust::raw_pointer_cast (
                  mask.data());
            cu_real_t * d_stat = thrust::raw_pointer_cast (
                  stat.data());
            cu_real_t * d_accu = thrust::raw_pointer_cast (
                  mean_stat.data());

            cu_real_t * d_x_spin = thrust::raw_pointer_cast(
                  cu::atoms::x_spin_array.data());
            cu_real_t * d_y_spin = thrust::raw_pointer_cast(
                  cu::atoms::y_spin_array.data());
            cu_real_t * d_z_spin = thrust::raw_pointer_cast(
                  cu::atoms::z_spin_array.data());
            cu_real_t * d_spin_norm = thrust::raw_pointer_cast(
                  cu::atoms::spin_norm_array.data());

            int n_bins = stat.size ();
            int n_atoms = mask.size ();

            if (n_bins < 128)
            {

                // Use the shared memory implementation

               int n_bytes = 4 * stat.size() * sizeof(cu_real_array_t::value_type);
               hist_by_key_small_mask <<< cu::grid_size, cu::block_size, n_bytes >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
               check_cuda_errors (__FILE__, __LINE__);
            }
            else
            {

               // Use the brute force implementation

               hist_by_key_big_mask <<< cu::grid_size, cu::block_size >>> (
                     d_x_spin,
                     d_y_spin,
                     d_z_spin,
                     d_spin_norm,
                     d_mask,
                     d_stat,
                     n_bins,
                     n_atoms
                     );
               check_cuda_errors (__FILE__, __LINE__);
            }


             // Reduce and accumulate

            int gs = n_bins / cu::block_size + 1;
            update_norm_and_accum <<< gs , cu::block_size >>> (
                  d_stat,
                  d_accu,
                  n_bins
                  );
            check_cuda_errors (__FILE__, __LINE__);

         }


         void __get_stat (
               const cu_real_array_t& stat,
               const cu_real_array_t& mean_stat,
               ::stats::magnetization_statistic_t& local_stat
               )
         {

            /*
             * Copy to local arrays
             */

            thrust::host_vector<cu_real_t> h_stat(stat.size());
            thrust::host_vector<cu_real_t> h_mean_stat(mean_stat.size());

            thrust::copy(stat.begin(), stat.end(), h_stat.begin());
            thrust::copy(mean_stat.begin(), mean_stat.end(), h_mean_stat.begin());

            /*
             * Call the method in the magnetization_statistic_t instance
             */

            std::vector<cu_real_t> stl_stat (h_stat.begin(), h_stat.end());
            std::vector<cu_real_t> stl_mean_stat (h_mean_stat.begin(), h_mean_stat.end());

            local_stat.set_magnetization (
                  stl_stat,
                  stl_mean_stat,
                  counter);
            check_cuda_errors (__FILE__, __LINE__);

         }


         void __reset_stat (
               cu_real_array_t& stat,
               cu_real_array_t& mean_stat
               )
         {
            thrust::fill(
                  stat.begin(),
                  stat.end(),
                  0.0);
            thrust::fill(
                  mean_stat.begin(),
                  mean_stat.end(),
                  0.0);
            check_cuda_errors (__FILE__, __LINE__);
         }


         __global__ void hist_by_key_small_mask (
               const cu_real_t * __restrict__ x_spin,
               const cu_real_t * __restrict__ y_spin,
               const cu_real_t * __restrict__ z_spin,
               const cu_real_t * __restrict__ norm_spin,
               const int * __restrict__ mask,
               cu_real_t * hist,
               int n_bins,
               int n_atoms
               )
         {
            extern __shared__ cu_real_t block_hist[];

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Initialize block memory
                */
               block_hist[i] = 0.0;
            }

            __syncthreads ();

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {
               /*
                * Store stuff in the shared memory
                */
               int bin = mask[i];
               cu_real_t mu_s = norm_spin[i];
               cu::atomicAdd (block_hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 3, mu_s);
            }

            __syncthreads ();

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Store stuff in the main memory
                */
               cu::atomicAdd (hist + 4 * i + 0, block_hist[4 * i + 0]);
               cu::atomicAdd (hist + 4 * i + 1, block_hist[4 * i + 1]);
               cu::atomicAdd (hist + 4 * i + 2, block_hist[4 * i + 2]);
               cu::atomicAdd (hist + 4 * i + 3, block_hist[4 * i + 3]);
            }

         }


         __global__ void hist_by_key_big_mask (
               const cu_real_t * __restrict__ x_spin,
               const cu_real_t * __restrict__ y_spin,
               const cu_real_t * __restrict__ z_spin,
               const cu_real_t * __restrict__ norm_spin,
               const int * __restrict__ mask,
               cu_real_t * hist,
               int n_bins,
               int n_atoms
               )
         {

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {

                // Store stuff in the main memory

               int bin = mask[i];
               cu_real_t mu_s = norm_spin[i];
               cu::atomicAdd (hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 3, mu_s);
            }
         }


         __global__ void update_norm_and_accum (
               cu_real_t * hist,
               cu_real_t * accum,
               int n_bins
               )
         {

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_bins;
                  i += blockDim.x * gridDim.x)
            {
               cu_real_t mx = hist[4 * i + 0];
               cu_real_t my = hist[4 * i + 1];
               cu_real_t mz = hist[4 * i + 2];
               cu_real_t ms = hist[4 * i + 3];

               cu_real_t mm = sqrtf (
                     mx * mx +
                     my * my +
                     mz * mz
                     );

               hist[4 * i + 0] = mx / mm;
               hist[4 * i + 1] = my / mm;
               hist[4 * i + 2] = mz / mm;
               hist[4 * i + 3] = mm / ms;

               accum[4 * i + 0] += mx / mm;
               accum[4 * i + 1] += my / mm;
               accum[4 * i + 2] += mz / mm;
               accum[4 * i + 3] += mm / ms;
            }
         }

      } /* stats */
   } /* internal */

#endif

} // end of namespace cuda
