#include "hip/hip_runtime.h"
//-----------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) R F L Evans 2015. All rights reserved.
//
//-----------------------------------------------------------------------------

// C++ standard library headers
#include <thrust/sort.h>

// Vampire headers
#include "cuda.hpp"

// Local cuda headers
#include "internal.hpp"
#include "statistics.hpp"

#ifdef CUDA
namespace cu = vcuda::internal;
#endif

namespace vcuda{

   //-------------------------------------------------------------------------------
   // Function to update statistics
   //-------------------------------------------------------------------------------
   void stats_update(){

      #ifdef CUDA


      #endif

      return;
   }

#ifdef CUDA

   namespace internal
   {
      namespace stats
      {

         void __update_stat (
               const IndexArray & mask,
               const RealArray & stat_saturation,
               RealArray& stat,
               RealArray& mean_stat
               )
         {

         }


         __global__ void hist_by_key_small_mask (
               const double * __restrict__ x_spin,
               const double * __restrict__ y_spin,
               const double * __restrict__ z_spin,
               const double * __restrict__ norm_spin,
               const int * __restrict__ mask,
               double * hist,
               int n_bins,
               int n_atoms
               )
         {
            extern __shared__ double block_hist[];

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Initialize block memory
                */
               block_hist[i] = 0.0;
            }

            __syncthreads ();

            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {
               /*
                * Store stuff in the shared memory
                */
               int bin = mask[i];
               double mu_s = norm_spin[i];
               cu::atomicAdd (block_hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (block_hist + 4 * bin + 3, mu_s);
            }

            __syncthreads ();

            for (int i = threadIdx.x; i < 4 * n_bins; i += blockDim.x)
            {
               /*
                * Store stuff in the main memory
                */
               cu::atomicAdd (hist + 4 * i + 0, block_hist[4 * i + 0]);
               cu::atomicAdd (hist + 4 * i + 1, block_hist[4 * i + 1]);
               cu::atomicAdd (hist + 4 * i + 2, block_hist[4 * i + 2]);
               cu::atomicAdd (hist + 4 * i + 3, block_hist[4 * i + 3]);
            }

         }


         __global__ void hist_by_key_big_mask (
               const double * __restrict__ x_spin,
               const double * __restrict__ y_spin,
               const double * __restrict__ z_spin,
               const double * __restrict__ norm_spin,
               const int * __restrict__ mask,
               double * hist,
               int n_bins,
               int n_atoms
               )
         {
            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_atoms;
                  i += blockDim.x * gridDim.x)
            {
               /*
                * Store stuff in the main memory
                */
               int bin = mask[i];
               double mu_s = norm_spin[i];
               cu::atomicAdd (hist + 4 * bin + 0, x_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 1, y_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 2, z_spin[i] * mu_s);
               cu::atomicAdd (hist + 4 * bin + 3, mu_s);
            }
         }


         __global__ void update_norm_and_accum (
               double * hist,
               double * accum,
               int n_bins
               )
         {
            for ( int i = blockIdx.x * blockDim.x + threadIdx.x;
                  i < n_bins;
                  i += blockDim.x * gridDim.x)
            {
               double mx = hist[4 * i + 0];
               double my = hist[4 * i + 1];
               double mz = hist[4 * i + 2];
               double ms = hist[4 * i + 3];

               double mm = sqrtf (
                     mx * mx +
                     my * my +
                     mz * mz
                     );

               hist[4 * i + 0] = mx / mm;
               hist[4 * i + 1] = my / mm;
               hist[4 * i + 2] = mz / mm;
               hist[4 * i + 3] = mm / ms;

               accum[4 * i + 0] += mx / mm;
               accum[4 * i + 1] += my / mm;
               accum[4 * i + 2] += mz / mm;
               accum[4 * i + 3] += mm / ms;
            }
         }

      } /* stats */
   } /* internal */

#endif

} // end of namespace cuda
