#include "hip/hip_runtime.h"
#include "internal.hpp"

#ifdef CUDA
namespace cu = vcuda::internal;
#endif

namespace cuda
{
#ifdef CUDA
   namespace internal
   {

      __global__ void update_non_exchange_spin_fields (
            double * x_spin, double * y_spin, double * z_spin,
            size_t * material,
            vcuda::internal::material_parameters_t * material_params,
            double * x_sp_field, double * y_sp_field, double * z_sp_field,
            size_t n_atoms
            )
      {
         for ( size_t i = blockIdx.x * blockDim.x + threadIdx.x;
               i < n_atoms;
               i += blockDim.x * gridDim.x)
         {

            size_t mid = material[i];

            double field_x = 0.0;
            double field_y = 0.0;
            double field_z = 0.0;

            double sx = x_spin[i];
            double sy = y_spin[i];
            double sz = z_spin[i];

            /*
             * Scalar anisotropy
             */
            double ku = material_params[mid].ku;
            field_z -= 2.0 * ku * sz;

            /*
             * Second order uniaxial anisotropy
             */

            double ku2 = 4.0 * material_params[mid].ku2;

            double ex = material_params[mid].anisotropy_unit_x;
            double ey = material_params[mid].anisotropy_unit_y;
            double ez = material_params[mid].anisotropy_unit_z;

            double sdote = sx * ex + sy * ey + sz * ez;
            double sdote3 = sdote * sdote * sdote;
            field_x -= ku2 * ex * sdote3;
            field_y -= ku2 * ey * sdote3;
            field_z -= ku2 * ez * sdote3;

            /*
             * Sixth order o¿uniaxial anisotropy
             */

            double ku3 = 6.0 * material_params[mid].ku3;
            double sdote5 = sdote3 * sdote * sdote;
            field_x -= ku3 * ex * sdote5;
            field_y -= ku3 * ey * sdote5;
            field_z -= ku3 * ez * sdote5;

            /*
             * Spherical harmonics
             */

            double scale = 0.6666666666666667;

            double mu_s_si = material_params[mid].mu_s_SI;
            double k2 = material_params[mid].sh2 / mu_s_si;
            double k4 = material_params[mid].sh4 / mu_s_si;
            double k6 = material_params[mid].sh6 / mu_s_si;

            double ek2 = k2 * 3.0 * sdote;
            double ek4 = k4 * 0.125 * (140.0 * sdote3 - 60.0 *sdote);
            double ek6 = k6 * 0.0625 * (
                  1386.0 * sdote5 - 1260.0 * sdote3 + 210.0 * sdote);

            field_x += scale * ex * (ek2 + ek4 + ek6);
            field_y += scale * ey * (ek2 + ek4 + ek6);
            field_z += scale * ez * (ek2 + ek4 + ek6);

            /*
             * Lattice anisotropy
             * TODO: add the temperature dependence
             */

            double k_latt = 2.0 * material_params[mid].Klatt_SI / mu_s_si;
            field_x -= k_latt * ex * sdote;
            field_y -= k_latt * ey * sdote;
            field_z -= k_latt * ez * sdote;

            /*
             * TODO: Surface anisotropy?
             */

            /*
             * TODO: Lagrange multipliers?
             */

            /*
             * Write back to main memory
             */

            x_sp_field[i] += field_x;
            y_sp_field[i] += field_y;
            z_sp_field[i] += field_z;
         }
      }

      __global__ void update_external_fields (
            size_t * material, size_t * cell,
            vcuda::internal::material_parameters_t * material_params,
            double * x_dip_field, double * y_dip_field, double * z_dip_field,
            double * x_ext_field, double * y_ext_field, double * z_ext_field,
            hiprandState * rand_state, size_t n_atoms
            )
      {

         /*
          * Thread and material identification
          */

         size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

         for ( size_t i = tid;
               i < n_atoms;
               i += blockDim.x * gridDim.x)
         {

            size_t mid = material[i];
            cu::material_parameters_t mat = material_params[mid];

            double field_x = 0.0;
            double field_y = 0.0;
            double field_z = 0.0;

            /*
             * TODO: HAMR fields
             */

            /*
             * Thermal fields
             */

            double temp = mat.temperature;
            double alpha = mat.temperature_rescaling_alpha;
            double sigma = mat.H_th_sigma;
            double tc = mat.temperature_rescaling_Tc;
            double resc_temp = (temp < tc) ? tc * pow(temp / tc, alpha) : temp;
            double sq_temp = sqrt(resc_temp);

            field_x += sigma * sq_temp * hiprand_normal_double (
                  rand_state + tid);
            field_y += sigma * sq_temp * hiprand_normal_double (
                  rand_state + tid);
            field_z += sigma * sq_temp * hiprand_normal_double (
                  rand_state + tid);

            /*
             * Applied field
             */

            double norm_h = mat.applied_field_strength;
            double hx = mat.applied_field_unit_x;
            double hy = mat.applied_field_unit_y;
            double hz = mat.applied_field_unit_z;

            field_x += norm_h * hx;
            field_y += norm_h * hy;
            field_z += norm_h * hz;

            /*
             * TODO: FMR fields?
             */

            /*
             * Dipolar fields
             */

            field_x += x_dip_field[i];
            field_y += y_dip_field[i];
            field_z += z_dip_field[i];

            /*
             * Write back to main memory
             */

            x_ext_field[i] += field_x;
            y_ext_field[i] += field_y;
            z_ext_field[i] += field_z;

         }
      }

      __global__ void update_cell_magnetization (
            double * x_spin, double * y_spin, double * z_spin,
            size_t * material, size_t * cell,
            vcuda::internal::material_parameters_t * material_params,
            double * x_mag, double * y_mag, double * z_mag,
            size_t n_atoms
            )
      {
         /*
          * TODO: This is an supremely naïve implementation
          *       the number of cells can be as big as the number of atoms
          *       so might as well leave it like this
          */

         for ( size_t i = blockIdx.x * blockDim.x + threadIdx.x;
               i < n_atoms;
               i += blockDim.x * gridDim.x)
         {
            size_t mid = material[i];
            size_t cid = cell[i];
            double mu_s = material_params[mid].mu_s_SI;
            cu::atomicAdd(&x_mag[cid], x_spin[i] * mu_s);
            cu::atomicAdd(&y_mag[cid], y_spin[i] * mu_s);
            cu::atomicAdd(&z_mag[cid], z_spin[i] * mu_s);
         }
      }

      __global__ void update_dipolar_fields (
            double * x_mag, double * y_mag, double * z_mag,
            double * x_coord, double * y_coord, double * z_coord,
            double * volume, double prefactor,
            double * x_dip_field, double * y_dip_field, double * z_dip_field,
            size_t n_cells
            )
      {
         for ( size_t i = blockIdx.x * blockDim.x + threadIdx.x;
               i < n_cells;
               i += blockDim.x * gridDim.x)
         {
            double mx = x_mag[i];
            double my = y_mag[i];
            double mz = z_mag[i];
            double cx = x_coord[i];
            double cy = y_coord[i];
            double cz = z_coord[i];
            /*
             * Inverse volume from the number of atoms in macro-cell
             */
            double vol_prefac = - 4.0 * M_PI / (3.0 * volume[i]);

            double field_x = vol_prefac * mx;
            double field_y = vol_prefac * my;
            double field_z = vol_prefac * mz;

            for (size_t j = 0; j < n_cells; j++)
            {
               if (i == j) continue;
               double omx = x_mag[i];
               double omy = y_mag[i];
               double omz = z_mag[i];

               double dx = x_coord[j] - cx;
               double dy = y_coord[j] - cy;
               double dz = z_coord[j] - cz;

               double drij = 1.0 / sqrtf (dx * dx + dy * dy + dz * dz);
               double drij3 = drij * drij * drij;

               double sdote = (
                     omx * dx * drij +
                     omy * dy * drij +
                     omz * dz * drij);

               field_x += (3.0 * sdote * dx * drij - omx) * drij3;
               field_y += (3.0 * sdote * dy * drij - omy) * drij3;
               field_z += (3.0 * sdote * dz * drij - omz) * drij3;
            }

            x_dip_field[i] = prefactor * field_x;
            y_dip_field[i] = prefactor * field_y;
            z_dip_field[i] = prefactor * field_z;
         }
      }

   } /* internal */
#endif
} /* cuda */

