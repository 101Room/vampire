#include "hip/hip_runtime.h"
/**
 * @brief this file provides definitions for the off-topic internal function
 *        definitions.
 */

#include "data.hpp"
#include "internal.hpp"

#ifdef CUDA
namespace cu = ::vcuda::internal;
#endif

namespace vcuda
{
#ifdef CUDA

   namespace internal
   {

      size_t block_size(256UL);
      size_t grid_size(512UL);

      __global__ void init_rng (hiprandState * state, size_t seed)
      {
         size_t tid = blockIdx.x + blockDim.x + threadIdx.x;
         hiprand_init (seed, tid, 0, state + tid);
      }
   }

#endif
} /* vcuda */
